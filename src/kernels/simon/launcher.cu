/**
 * @file launcher.cu
 * @brief Entry point for launching kernels
 * 
 */

#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cmath>

#include "utils.h"
#include "kernel01_naive.cuh"

namespace simon
{
    /**
     * @brief Launches a naive sgemm kernel
     * 
     * @param A       Pointer to input matrix A, stored in row-major order
     * @param B       Pointer to input matrix B
     * @param C       Pointer to output matrix C
     * @param M       Number of rows in matrix A and C
     * @param N       Number of columns in A and rows in B (shared dimension)
     * @param K       Number of columns in matrices B and C
     * @param alpha   Scalar multiplier for the matrix product (A @ B)
     * @param beta    Scalar multiplier for the existing values in matrix C
     */
    void run_sgemm_naive(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C, 
        int M, int N, int K, float alpha, float beta) {
            // Grid configs
            dim3 gridDim(CEIL_DIV(K, 32), CEIL_DIV(M, 32));
            dim3 blockDim(32, 32);
            sgemm_naive<<<gridDim, blockDim>>>(A, B, C, M, N, K, alpha, beta);
            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipDeviceSynchronize());
        }
}