/**
 * @file launcher.cu
 * @brief Entry point for launching kernels
 * 
 */

#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cmath>

#include "utils.h"
#include "simon_naive.cuh"
#include "simon_coalesced.cuh"

namespace simon {
    /**
     * @brief Launches a naive sgemm kernel
     * 
     * @param A       Pointer to input matrix A, stored in row-major order
     * @param B       Pointer to input matrix B
     * @param C       Pointer to output matrix C
     * @param M       Number of rows in matrix A and C
     * @param N       Number of columns in A and rows in B (shared dimension)
     * @param K       Number of columns in matrices B and C
     * @param alpha   Scalar multiplier for the matrix product (A @ B)
     * @param beta    Scalar multiplier for the existing values in matrix C
     */
    void run_sgemm_naive(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C, 
        int M, int N, int K, float alpha, float beta) {
            // Grid configs
            dim3 gridDim(CEIL_DIV(K, 32), CEIL_DIV(M, 32));
            dim3 blockDim(32, 32);
            sgemm_naive<<<gridDim, blockDim>>>(A, B, C, M, N, K, alpha, beta);
            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipDeviceSynchronize());
        }

    void run_sgemm_coalesced(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C,
        int M, int N, int K, float alpha, float beta) {
            dim3 gridDim(CEIL_DIV(K, 32), CEIL_DIV(M, 32));
            dim3 blockDim(32*32);
            sgemm_coalesced<32><<<gridDim, blockDim>>>(A, B, C, M, N, K, alpha, beta);
            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipDeviceSynchronize());
        }
}

namespace cublas {
    /**
 * @brief Launches the cuBLAS SGEMM reference kernel using hipblasGemmEx.
 *
 * This function uses cuBLAS to perform C = alpha * A * B + beta * C.
 * All matrices are assumed to be in row-major order.
 * cuBLAS expects column-major, so we swap A/B and M/N.
 *
 * @param A       Pointer to input matrix A (device, row-major)
 * @param B       Pointer to input matrix B (device, row-major)
 * @param C       Pointer to output matrix C (device, row-major)
 * @param M       Number of rows of matrix A and C
 * @param N       Number of columns of matrix B and C
 * @param K       Number of columns of matrix A and rows of matrix B
 * @param alpha   Scalar multiplier for the matrix product
 * @param beta    Scalar multiplier for the existing values in matrix C
 * @param handle  cuBLAS handle
 */
void run_sgemm_cublas(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C,
                      int M, int N, int K, float alpha, float beta, hipblasHandle_t handle) {
    // cuBLAS uses column-major order. So we change the order of our row-major A & B,
    // since (B^T*A^T)^T = (A*B)
    hipblasStatus_t stat = hipblasGemmEx(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        N, M, K, // Note: N and M are swapped for row-major
        &alpha,
        B, HIP_R_32F, N, // B is on the left in column-major
        A, HIP_R_32F, K,
        &beta,
        C, HIP_R_32F, N,
        HIPBLAS_COMPUTE_32F,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP
    );
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "cuBLAS hipblasGemmEx failed!" << std::endl;
    }
    CUDA_CHECK(hipDeviceSynchronize());
}
}