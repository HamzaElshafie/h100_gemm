/**
 * @file launcher.cu
 * @brief Entry point for launching kernels
 * 
 */

#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cmath>

#include "utils.h"
#include "sgemm_naive.cuh"
#include "sgemm_coalesced.cuh"
#include "sgemm_tiled_shared.cuh"
#include "sgemm_1D_registertiling.cuh"
#include "sgemm_2D_registertiling.cuh"

namespace ampere {
    /**
     * @brief Launches a naive sgemm kernel
     * 
     * @param A       Pointer to input matrix A, stored in row-major order
     * @param B       Pointer to input matrix B
     * @param C       Pointer to output matrix C
     * @param M       Number of rows in matrix A and C
     * @param N       Number of columns in A and rows in B (shared dimension)
     * @param K       Number of columns in matrices B and C
     * @param alpha   Scalar multiplier for the matrix product (A @ B)
     * @param beta    Scalar multiplier for the existing values in matrix C
     */
    void run_sgemm_naive(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C, 
        int M, int N, int K, float alpha, float beta) {
            // Grid configs
            dim3 gridDim(CEIL_DIV(K, 32), CEIL_DIV(M, 32));
            dim3 blockDim(32, 32);
            sgemm_naive<<<gridDim, blockDim>>>(A, B, C, M, N, K, alpha, beta);
            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipDeviceSynchronize());
        }

    /**
     * @brief Launches a coalesced sgemm kernel
     * 
     * @param A       Pointer to input matrix A, stored in row-major order
     * @param B       Pointer to input matrix B
     * @param C       Pointer to output matrix C
     * @param M       Number of rows in matrix A and C
     * @param N       Number of columns in A and rows in B (shared dimension)
     * @param K       Number of columns in matrices B and C
     * @param alpha   Scalar multiplier for the matrix product (A @ B)
     * @param beta    Scalar multiplier for the existing values in matrix C
     */
    void run_sgemm_coalesced(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C,
        int M, int N, int K, float alpha, float beta) {
            dim3 gridDim(CEIL_DIV(K, 32), CEIL_DIV(M, 32));
            dim3 blockDim(32*32); // 1024 threads per block
            sgemm_coalesced<32><<<gridDim, blockDim>>>(A, B, C, M, N, K, alpha, beta);
            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipDeviceSynchronize());
        }

    void run_sgemm_tiled_shared(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C,
    int M, int N, int K, float alpha, float beta) {
        dim3 gridDim(CEIL_DIV(K, 32), CEIL_DIV(M, 32));
        dim3 blockDim(32*32); // 1024 threads per block
        sgemm_tiled_shared<32><<<gridDim, blockDim>>>(A, B, C, M, N, K, alpha, beta);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
    }

    void run_sgemm_1D_registertiling(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C,
    int M, int N, int K, float alpha, float beta) {
        const uint TILE_SIZE_M = 64;
        const uint TILE_SIZE_K = 64;
        const uint TILE_SIZE_N = 8;
        const uint ROWS_PER_THREAD = 8;
        dim3 gridDim(CEIL_DIV(K, TILE_SIZE_K), CEIL_DIV(M, TILE_SIZE_M));
        dim3 blockDim((TILE_SIZE_M * TILE_SIZE_K) / ROWS_PER_THREAD);
        sgemm_1D_registertiling<TILE_SIZE_M, TILE_SIZE_N, TILE_SIZE_K, ROWS_PER_THREAD>
            <<<gridDim, blockDim>>>(A, B, C, M, N, K, alpha, beta);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
    }

    void run_sgemm_2D_registertiling(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C,
                                 int M, int N, int K, float alpha, float beta) {
    const uint TILE_SIZE_N = 8;
    const uint ROWS_PER_THREAD = 8;
    const uint COLS_PER_THREAD = 8;

    if (M >= 128 && K >= 128) {
        const uint TILE_SIZE_M = 128;
        const uint TILE_SIZE_K = 128;
        dim3 gridDim(CEIL_DIV(K, TILE_SIZE_K), CEIL_DIV(M, TILE_SIZE_M));
        dim3 blockDim((TILE_SIZE_M * TILE_SIZE_K) / (ROWS_PER_THREAD * COLS_PER_THREAD));
        sgemm_2D_registertiling<TILE_SIZE_M, TILE_SIZE_N, TILE_SIZE_K, ROWS_PER_THREAD, COLS_PER_THREAD>
            <<<gridDim, blockDim>>>(A, B, C, M, N, K, alpha, beta);
    } else {
        // fallback to smaller tile
        const uint TILE_SIZE_M = 64;
        const uint TILE_SIZE_K = 64;
        dim3 gridDim(CEIL_DIV(K, TILE_SIZE_K), CEIL_DIV(M, TILE_SIZE_M));
        dim3 blockDim((TILE_SIZE_M * TILE_SIZE_K) / (ROWS_PER_THREAD * COLS_PER_THREAD));
        sgemm_2D_registertiling<TILE_SIZE_M, TILE_SIZE_N, TILE_SIZE_K, ROWS_PER_THREAD, COLS_PER_THREAD>
            <<<gridDim, blockDim>>>(A, B, C, M, N, K, alpha, beta);
    }
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    }
}

namespace cublas {

    void run_sgemm_cublas(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C,
                      int M, int N, int K, float alpha, float beta,
                      hipblasHandle_t handle) {

    const void* alpha_ptr = static_cast<const void*>(&alpha);
    const void* beta_ptr  = static_cast<const void*>(&beta);

    //hipblasComputeType_t computeType = HIPBLAS_COMPUTE_32F;             // Tensor Cores on 
    hipblasComputeType_t computeType = HIPBLAS_COMPUTE_32F_PEDANTIC; // Tensor Cores off

    CUBLAS_CHECK(hipblasGemmEx(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        K, M, N,
        alpha_ptr,
        B, HIP_R_16F, K,
        A, HIP_R_16F, N,
        beta_ptr,
        C, HIP_R_32F, K,
        computeType,
        HIPBLAS_GEMM_DEFAULT
    ));

    CUDA_CHECK(hipDeviceSynchronize());
}
}