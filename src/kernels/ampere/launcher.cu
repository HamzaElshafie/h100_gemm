/**
 * @file launcher.cu
 * @brief Entry point for launching kernels
 * 
 */

#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cmath>

#include "utils.h"
#include "sgemm_naive.cuh"
#include "sgemm_coalesced.cuh"
#include "sgemm_tiled_shared.cuh"

namespace ampere {
    /**
     * @brief Launches a naive sgemm kernel
     * 
     * @param A       Pointer to input matrix A, stored in row-major order
     * @param B       Pointer to input matrix B
     * @param C       Pointer to output matrix C
     * @param M       Number of rows in matrix A and C
     * @param N       Number of columns in A and rows in B (shared dimension)
     * @param K       Number of columns in matrices B and C
     * @param alpha   Scalar multiplier for the matrix product (A @ B)
     * @param beta    Scalar multiplier for the existing values in matrix C
     */
    void run_sgemm_naive(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C, 
        int M, int N, int K, float alpha, float beta) {
            // Grid configs
            dim3 gridDim(CEIL_DIV(K, 32), CEIL_DIV(M, 32));
            dim3 blockDim(32, 32);
            sgemm_naive<<<gridDim, blockDim>>>(A, B, C, M, N, K, alpha, beta);
            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipDeviceSynchronize());
        }

    /**
     * @brief Launches a coalesced sgemm kernel
     * 
     * @param A       Pointer to input matrix A, stored in row-major order
     * @param B       Pointer to input matrix B
     * @param C       Pointer to output matrix C
     * @param M       Number of rows in matrix A and C
     * @param N       Number of columns in A and rows in B (shared dimension)
     * @param K       Number of columns in matrices B and C
     * @param alpha   Scalar multiplier for the matrix product (A @ B)
     * @param beta    Scalar multiplier for the existing values in matrix C
     */
    void run_sgemm_coalesced(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C,
        int M, int N, int K, float alpha, float beta) {
            dim3 gridDim(CEIL_DIV(K, 32), CEIL_DIV(M, 32));
            dim3 blockDim(32*32); // 1024 threads per block
            sgemm_coalesced<32><<<gridDim, blockDim>>>(A, B, C, M, N, K, alpha, beta);
            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipDeviceSynchronize());
        }

    void run_sgemm_tiled_shared(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C,
    int M, int N, int K, float alpha, float beta) {
        dim3 gridDim(CEIL_DIV(K, 32), CEIL_DIV(M, 32));
        dim3 blockDim(32*32); // 1024 threads per block
        sgemm_tiled_shared<32><<<gridDim, blockDim>>>(A, B, C, M, N, K, alpha, beta);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
    }
}

namespace cublas {
    /**
 * @brief Launches the cuBLAS SGEMM reference kernel using hipblasGemmEx.
 *
 * This function uses cuBLAS to perform C = alpha * A * B + beta * C.
 * All matrices are assumed to be in row-major order.
 * cuBLAS expects column-major, so we swap A/B and M/N.
 *
 * @param A       Pointer to input matrix A (device, row-major)
 * @param B       Pointer to input matrix B (device, row-major)
 * @param C       Pointer to output matrix C (device, row-major)
 * @param M       Number of rows of matrix A and C
 * @param N       Number of columns of matrix B and C
 * @param K       Number of columns of matrix A and rows of matrix B
 * @param alpha   Scalar multiplier for the matrix product
 * @param beta    Scalar multiplier for the existing values in matrix C
 * @param handle  cuBLAS handle
 */
void run_sgemm_cublasLt(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C,
    int M, int N, int K, float alpha, float beta, hipblasHandle_t /*unused*/) {

    // Create (once) cuBLASLt handle & workspace
    static hipblasLtHandle_t ltHandle = nullptr;
    static void* workspace         = nullptr;
    static const size_t workspace_size = 32 * 1024 * 1024; // 32 MB

    if (ltHandle == nullptr) {
        CUBLAS_CHECK(hipblasLtCreate(&ltHandle));
        CUDA_CHECK(hipMalloc(&workspace, workspace_size));
    }

    // --- Create operation descriptor (compute in FP32) ---
    hipblasLtMatmulDesc_t operationDesc;
    CUBLAS_CHECK(hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));

    hipblasOperation_t opN = HIPBLAS_OP_N;
    CUBLAS_CHECK(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &opN, sizeof(opN)));
    CUBLAS_CHECK(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &opN, sizeof(opN)));

    // Scaling type (alpha / beta)
    hipDataType scale_type = HIP_R_32F;
    CUBLAS_CHECK(hipblasLtMatmulDescSetAttribute(operationDesc,
        CUBLASLT_MATMUL_DESC_SCALE_TYPE, &scale_type, sizeof(scale_type)));

    // --- Create matrix layouts (row-major order) ---
    hipblasLtMatrixLayout_t ALayout, BLayout, CLayout;
    CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&ALayout, HIP_R_32F, M, N, N));
    CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&BLayout, HIP_R_32F, N, K, K));
    CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&CLayout, HIP_R_32F, M, K, K));

    int32_t order = HIPBLASLT_ORDER_ROW;
    CUBLAS_CHECK(hipblasLtMatrixLayoutSetAttribute(ALayout,
        HIPBLASLT_MATRIX_LAYOUT_ORDER, &order, sizeof(order)));
    CUBLAS_CHECK(hipblasLtMatrixLayoutSetAttribute(BLayout,
        HIPBLASLT_MATRIX_LAYOUT_ORDER, &order, sizeof(order)));
    CUBLAS_CHECK(hipblasLtMatrixLayoutSetAttribute(CLayout,
        HIPBLASLT_MATRIX_LAYOUT_ORDER, &order, sizeof(order)));

    // --- Set preference & select heuristic algorithm ---
    hipblasLtMatmulPreference_t preference;
    CUBLAS_CHECK(hipblasLtMatmulPreferenceCreate(&preference));
    CUBLAS_CHECK(hipblasLtMatmulPreferenceSetAttribute(preference, CUBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTE, 
        &workspace_size, sizeof(workspace_size)));

    hipblasLtMatmulHeuristicResult_t heuristicResult;
    int returnedResults = 0;
    hipblasLtMatmulAlgoGetHeuristic(
        ltHandle,
        operationDesc,
        ALayout, BLayout, CLayout, CLayout,
        preference,
        1,
        &heuristicResult,
        &returnedResults);

    if (returnedResults == 0) {
        std::cerr << "cuBLASLt: No suitable algorithm found for (M,N,K) = ("
                  << M << ", " << N << ", " << K << ")" << std::endl;
        exit(EXIT_FAILURE);
    }

    // --- Perform GEMM ---
    CUBLAS_CHECK(hipblasLtMatmul(ltHandle,
                                operationDesc,
                                &alpha,
                                A, ALayout,
                                B, BLayout,
                                &beta,
                                C, CLayout,
                                C, CLayout,
                                &heuristicResult.algo,
                                workspace,
                                workspace_size,
                                /* stream */ 0));

    // Cleanup descriptors (preference/layout/operation)
    CUBLAS_CHECK(hipblasLtMatmulPreferenceDestroy(preference));
    CUBLAS_CHECK(hipblasLtMatmulDescDestroy(operationDesc));
    CUBLAS_CHECK(hipblasLtMatrixLayoutDestroy(ALayout));
    CUBLAS_CHECK(hipblasLtMatrixLayoutDestroy(BLayout));
    CUBLAS_CHECK(hipblasLtMatrixLayoutDestroy(CLayout));

    CUDA_CHECK(hipDeviceSynchronize());
}
}