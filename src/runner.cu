/**
 * @file runner.cu
 * @brief Kernel launcher implementation.
 *
 * Dispatches the requested kernel to the launchers.
 */
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdexcept>

#include "runner.cuh"
#include "kernels/ampere/launcher.cuh"
#include "kernels/hopper/launcher.cuh"
