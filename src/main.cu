#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <vector>
#include <hipblas.h>

#include "utils.h"
#include "runner.cuh"

/**
 * @brief Prints the usage instructions for the program.
 *
 * This function outputs the correct CLI usage for the executable,
 * including the required arguments and an example.
 */
void printUsage() {
    std::cout << "Usage: ./sgemm <implementation> <kernel_ID_number>\n"
              << "  Implementation: simon | hopper | cublas\n"
              << "  ID:       0, 1, 2, ...\n" // TODO: Print last kernel number for each implementation
              << "Example: ./sgemm simon 0\n"
              << "(Note): To run cublas you must use ID=0. ./sgemm cublas 0\n";
}

/**
 * @brief Parses the kernel configuration from CLI arguments.
 *
 * @param impl      The implementation name (e.g., "simon" or "hopper").
 * @param kernel_id The kernel variant number.
 * @return KernelConfig The parsed kernel configuration.
 * @throws std::invalid_argument if the kernel_id is invalid.
 */
KernelConfig parseKernelConfig(const std::string& impl, int kernel_id) {
    if (impl == "simon") { // Check kernel validity
        if (kernel_id > 2 || kernel_id < 0) { // (TODO: Update later)
            throw std::invalid_argument("Invalid Simon kernel ID");
        }
        return KernelConfig(KernelType::SIMON, kernel_id);
    } else if (impl == "hopper") {
        if (kernel_id > 1 || kernel_id < 0) { // (TODO: Update later)
            throw std::invalid_argument("Invalid Hopper kernel ID");
        }
        return KernelConfig(KernelType::HOPPER, kernel_id);
    } else if (impl == "cublas") {
        if (kernel_id != 0) {
            throw std::invalid_argument("Invalid cuBLAS kernel ID"); // Only 0
        }
        return KernelConfig(KernelType::CUBLAS, kernel_id);
    } else {
        throw std::invalid_argument("Invalid implementation name: " + impl);
    }
}

/**
 * @brief Main entry point for the program.
 *
 * @param argc Number of CLI arguments.
 * @param argv Array of CLI strings.
 * @return int Exit status code.
 */
int main(int argc, char** argv) {
    ResourceManager resources;

    if (argc != 3) {
        printUsage();
        return -1;
    }
    
    std::string impl = argv[1];
    int kernel_id = std::stoi(argv[2]);
    KernelConfig config = parseKernelConfig(impl, kernel_id);
    
    // Define matrices sizes to test
    std::vector<int> sizes = {128, 256, 512, 1024, 2048, 4096, 8192};
    float alpha = 5.0f;
    float beta = 3.0f;

    // Calculate memory size required (Allocate for largest size and reuse for smaller matrices)
    int max_size = sizes.back();
    std::cout << "Max size: " << max_size << std::endl;
    size_t mem_size = max_size * max_size * sizeof(float);

    // Allocate host memory
    float* A_host = (float*)malloc(mem_size);
    float* B_host = (float*)malloc(mem_size);
    float* C_host = (float*)malloc(mem_size);
    float* C_host_ref = (float*)malloc(mem_size);

    // Register host memory with resource manager
    resources.add_host_ptr(A_host);
    resources.add_host_ptr(B_host);
    resources.add_host_ptr(C_host);
    resources.add_host_ptr(C_host_ref);

    if (!A_host || !B_host || !C_host || !C_host_ref) {
        std::cerr << "Host memory allocation failed" << std::endl;
        return -1;  // ResourceManager will clean up automatically
    }

    // Initialise matrices
    float* matrices[] = {A_host, B_host, C_host};
    initialiseArrays(matrices, 3, max_size * max_size, -100.0f, 100.0f, 0);

    // Allocate device memory
    float* A_device;
    float* B_device;
    float* C_device;
    float* C_device_ref;

    CUDA_CHECK(hipMalloc((void**)&A_device, mem_size));
    CUDA_CHECK(hipMalloc((void**)&B_device, mem_size));
    CUDA_CHECK(hipMalloc((void**)&C_device, mem_size));
    CUDA_CHECK(hipMalloc((void**)&C_device_ref, mem_size));

    // Register device memory
    resources.add_device_ptr(A_device);
    resources.add_device_ptr(B_device);
    resources.add_device_ptr(C_device);
    resources.add_device_ptr(C_device_ref);

    // Copy data from host to device
    CUDA_CHECK(hipMemcpy(A_device, A_host, mem_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(B_device, B_host, mem_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(C_device, C_host, mem_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(C_device_ref, C_host, mem_size, hipMemcpyHostToDevice));

    hipblasHandle_t handle;
    if (hipblasCreate(&handle)) {
        std::cerr << "Create cublas handle error." << std::endl;
        return -1;
    }
    resources.set_cublas_handle(&handle);

    // Create events to time trials
    hipEvent_t start;
    hipEvent_t stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    resources.add_event(start);
    resources.add_event(stop);
    float elapsed_time;
    
    int repeat = 50;
    for (int size: sizes) {
        int M = size;
        int N = size;
        int K = size;

        // Calculate current memory size required
        size_t curr_mem_size = size * size * sizeof(float);

        std::cout << "Dimensions (M = N = K) = " << M << " Alpha: " << alpha << " Beta: " << beta << std::endl;

        CUDA_CHECK(hipMemcpy(A_device, A_host, curr_mem_size, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(B_device, B_host, curr_mem_size, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(C_device, C_host, curr_mem_size, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(C_device_ref, C_host, curr_mem_size, hipMemcpyHostToDevice));

        // Run cuBLAS and custom kernel to check for correctness and warmup
        if (config.type != KernelType::CUBLAS) {
            // Custom op
            launchKernel(config, A_device, B_device, C_device, M, N, K, alpha, beta, handle);
            CUDA_CHECK(hipMemcpy(C_host, C_device, curr_mem_size, hipMemcpyDeviceToHost));
            // cuBLAS op
            KernelConfig cublas_config(KernelType::CUBLAS, 0);
            launchKernel(cublas_config, A_device, B_device, C_device_ref, M, N, K, alpha, beta, handle);
            CUDA_CHECK(hipMemcpy(C_host_ref, C_device_ref, curr_mem_size, hipMemcpyDeviceToHost));
            // Verify results
            bool results_match = compareResults(C_host_ref, C_host, M * K, 1e-1f, 1e-1f);
            if (!results_match) {
                std::cout << "Results do not match!" << std::endl;
                return -1;
            } else {
                std::cout << "Results match!" << std::endl;
            }
        }

        // Calculate total FLOPs for SGEMM: (2*M*N*K + 3*M*N) for alpha*(AB) + beta*C
        double flops_per_run = 2.0 * static_cast<double>(M) * static_cast<double>(N) * static_cast<double>(K) +
                              3.0 * static_cast<double>(M) * static_cast<double>(N);
        
        // Warmup cuBLAS kernel
        KernelConfig cublas_config(KernelType::CUBLAS, 0);
        launchKernel(cublas_config, A_device, B_device, C_device_ref, M, N, K, alpha, beta, handle);

        // Start cuBLAS timing
        CUDA_CHECK(hipEventRecord(start));
        // Run kernel multiple time to smooth out timing variations
        for (int i = 0; i < repeat; i++) {
            launchKernel(cublas_config, A_device, B_device, C_device, M, N, K, alpha, beta, handle);
        }
        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));
        CUDA_CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        
        elapsed_time /= 1000.; 
        double cublas_avg_time = elapsed_time / repeat;
        double cublas_gflops = flops_per_run / (cublas_avg_time * 1e9);

        // Start custom kernel timing step
        CUDA_CHECK(hipEventRecord(start));
        // Run kernel multiple time to smooth out timing variations
        for (int i = 0; i < repeat; i++) {
            launchKernel(config, A_device, B_device, C_device, M, N, K, alpha, beta, handle);
        }
        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));
        CUDA_CHECK(hipEventElapsedTime(&elapsed_time, start, stop));

        elapsed_time /= 1000.; // Convert to seconds
        double average_time = elapsed_time / repeat;
        // Throughput in GFLOPs/s
        double custom_gflops = flops_per_run / (average_time * 1e9);
        
        // Performance relative to cuBLAS
        double perf_ratio = custom_gflops / cublas_gflops;

        printf("Average elapsed time: %.6f s, GFLOPS: %.1f, Performance relative to cuBLAS: %.1f%%\n", average_time, custom_gflops, perf_ratio*100.0);
        
        // Copy result back to host
        CUDA_CHECK(hipMemcpy(C_host, C_device, curr_mem_size, hipMemcpyDeviceToHost));
    }

    return 0;  // ResourceManager destructor will all handle cleanups
}
